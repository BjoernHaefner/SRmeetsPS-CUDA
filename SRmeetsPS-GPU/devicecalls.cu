#include "hip/hip_runtime.h"
#include <devicecalls.cuh>
#include "Exceptions.h"

float* sort_COO(hipsparseHandle_t cusp_handle, int n_rows, int n_cols, int nnz, int* d_row_ind, int* d_col_ind, float* d_vals_unsorted) {
	hipsparseStatus_t status;
	float* d_vals = NULL;
	size_t pBufferSizeInBytes = 0;
	void *pBuffer = NULL;
	int *P = NULL;
	hipMalloc(&d_vals, nnz * sizeof(float)); CUDA_CHECK;
	status = hipsparseXcoosort_bufferSizeExt(cusp_handle, n_rows, n_cols, nnz, d_row_ind, d_col_ind, &pBufferSizeInBytes); CUSPARSE_CHECK(status);
	hipMalloc(&pBuffer, sizeof(char)* pBufferSizeInBytes); CUDA_CHECK;
	hipMalloc((void**)&P, sizeof(int)*nnz); CUDA_CHECK;
	status = hipsparseCreateIdentityPermutation(cusp_handle, nnz, P); CUSPARSE_CHECK(status);
	status = hipsparseXcoosortByRow(cusp_handle, n_rows, n_cols, nnz, d_row_ind, d_col_ind, P, pBuffer); CUSPARSE_CHECK(status);
	status = hipsparseSgthr(cusp_handle, nnz, d_vals_unsorted, d_vals, P, HIPSPARSE_INDEX_BASE_ZERO); CUSPARSE_CHECK(status);
	hipFree(d_vals_unsorted); CUDA_CHECK;
	hipFree(P); CUDA_CHECK;
	hipFree(pBuffer); CUDA_CHECK;
	return d_vals;
}

float* cuda_based_sparsemat_densevec_mul(hipsparseHandle_t& cusp_handle, int* row_ind, int* col_ind, float* vals, int n_rows, int n_cols, int nnz, float* d_vector) {
	int* d_row_ind = NULL;
	int* d_row_csr = NULL;
	int* d_col_ind = NULL;
	float* d_vals = NULL;
	float* d_output = NULL;
	hipMalloc(&d_col_ind, nnz * sizeof(int)); CUDA_CHECK;
	hipMalloc(&d_row_ind, nnz * sizeof(int)); CUDA_CHECK;
	hipMalloc(&d_vals, nnz * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_output, n_rows * sizeof(float)); CUDA_CHECK;
	hipMemcpy(d_row_ind, row_ind, nnz * sizeof(int), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(d_col_ind, col_ind, nnz * sizeof(int), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(d_vals, vals, nnz * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	d_vals = sort_COO(cusp_handle, n_rows, n_cols, nnz, d_row_ind, d_col_ind, d_vals);
	hipMalloc(&d_row_csr, (n_rows + 1) * sizeof(int)); CUDA_CHECK;
	hipsparseStatus_t cusp_stat;
	hipsparseMatDescr_t cusp_mat_desc = 0;
	cusp_stat = hipsparseCreateMatDescr(&cusp_mat_desc);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Matrix descriptor initialization failed");
	}
	hipsparseSetMatType(cusp_mat_desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(cusp_mat_desc, HIPSPARSE_INDEX_BASE_ZERO);

	cusp_stat = hipsparseXcoo2csr(cusp_handle, d_row_ind, (int)nnz, (int)n_rows, d_row_csr, HIPSPARSE_INDEX_BASE_ZERO);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Conversion from COO to CSR format failed");
	}
	float d_one = 1.f, d_zero = 0.f;
	cusp_stat = hipsparseScsrmv(cusp_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, (int)n_rows, (int)n_cols, (int)nnz, &d_one, cusp_mat_desc, d_vals, d_row_csr, d_col_ind, d_vector, &d_zero, d_output);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Matrix-vector multiplication failed");
	}
	cusp_stat = hipsparseDestroyMatDescr(cusp_mat_desc);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Matrix descriptor destruction failed");
	}

	hipFree(d_row_ind); CUDA_CHECK;
	hipFree(d_row_csr); CUDA_CHECK;
	hipFree(d_col_ind); CUDA_CHECK;
	hipFree(d_vals); CUDA_CHECK;
	return d_output;
}

__global__ void mean_across_channels(float* data, int h, int w, int nc, float* mean, uint8_t* inpaint_locations) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < h && j < w) {
		float avg = 0.f;
		for (int c = 0; c < nc; c++) {
			if (data[c*(w*h) + j*h + i] != 0)
				avg += data[c*(w*h) + j*h + i];
			else {
				inpaint_locations[j*h + i] = 1;
				//avg = NAN;
			}
		}
		mean[j*h + i] = avg / nc;
	}
}

float* cuda_based_mean_across_channels(float* data, int h, int w, int nc, uint8_t** d_inpaint_locations) {
	float* d_data = NULL;
	float* d_output = NULL;
	dim3 block(128, 8, 1);
	dim3 grid((unsigned)(h - 1) / block.x + 1, (unsigned)(w - 1) / block.y + 1, 1);
	hipMalloc(&d_data, h * w * nc * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_output, h * w * sizeof(float)); CUDA_CHECK;
	hipMalloc(d_inpaint_locations, h * w * sizeof(uint8_t)); CUDA_CHECK;
	hipMemset(*d_inpaint_locations, 0, h * w * sizeof(uint8_t)); CUDA_CHECK;
	hipMemcpy(d_data, data, h * w * nc * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	mean_across_channels << <grid, block >> > (d_data, h, w, nc, d_output, *d_inpaint_locations); CUDA_CHECK;
	hipFree(d_data); CUDA_CHECK;
	return d_output;
}

float* cuda_based_image_resize(float* data, int h, int w, int new_h, int new_w) {
	// TODO: switch to cv cuda
	return NULL;
}


__global__ void initialize_rho(float* rho, int size_c, int nc) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int c = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < size_c && c < nc) {
		rho[c*(size_c)+i] = 0.5f;
	}
}

float* cuda_based_rho_init(thrust::host_vector<int>& imask, int nc) {
	float* d_rho = NULL;
	hipMalloc(&d_rho, imask.size() * nc * sizeof(float)); CUDA_CHECK;
	dim3 block(512, 1, 1);
	dim3 grid((unsigned)(imask.size() - 1) / block.x + 1, (unsigned)(nc - 1) / block.y + 1, 1);
	initialize_rho << < grid, block >> > (d_rho, (int)imask.size(), nc); CUDA_CHECK;
	hipDeviceSynchronize();
	return d_rho;
}

__global__ void meshgrid_create(float* xx, float* yy, int w, int h) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < h && j < w) {
		xx[j*h + i] = j;
		yy[j*h + i] = i;
	}
}

std::pair<float*, float*> cuda_based_meshgrid_create(int w, int h) {
	float* xx = NULL, *yy = NULL;
	hipMalloc(&xx, sizeof(float)*w*h); CUDA_CHECK;
	hipMalloc(&yy, sizeof(float)*w*h); CUDA_CHECK;
	dim3 block(32, 8, 1);
	dim3 grid((unsigned)(w - 1) / block.x + 1, (unsigned)(h - 1) / block.y + 1, 1);
	meshgrid_create << <grid, block >> > (xx, yy, w, h);
	hipDeviceSynchronize();
	return std::pair<float*, float*>(xx, yy);
}

__global__ void third_and_fourth_normal_component(float* z, float* xx, float* yy, float* zx, float* zy, float K02, float K12, int npix, float* N3) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < npix) {
		N3[i] = -z[i] - (xx[i] - K02) * zx[i] - (yy[i] - K12) * zy[i];
		N3[npix + i] = 1;
	}
}

__global__ void norm_components(float* N, int npix, float* norm) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < npix) {
		norm[i] = fmaxf(1e-10, sqrtf(N[i] * N[i] + N[npix + i] * N[npix + i] + N[npix * 2 + i] * N[npix * 2 + i]));
	}
}

__global__ void normalize_N(float* N, float* norm, int npix_per_component) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int c = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < npix_per_component) {
		N[c*npix_per_component + i] = N[c*npix_per_component + i] / norm[i];
	}
}

float* cuda_based_normal_init(hipblasHandle_t cublas_handle, float *d_z, float* d_zx, float *d_zy, float *d_xx, float *d_yy, int npix, float K00, float K11, float K02, float K12) {
	float* d_N = NULL;
	float* d_norm = NULL;
	hipStream_t stream[3];
	hipMalloc(&d_N, sizeof(float)*npix * 4); CUDA_CHECK;
	hipMemset(d_N, 0, sizeof(float)*npix * 4); CUDA_CHECK;
	hipMalloc(&d_norm, sizeof(float)*npix); CUDA_CHECK;
	hipStreamCreate(&stream[0]); CUDA_CHECK;
	if (hipblasSetStream(cublas_handle, stream[0]) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	if (hipblasSaxpy(cublas_handle, npix, &K00, d_zx, 1, d_N, 1) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	hipStreamCreate(&stream[1]); CUDA_CHECK;
	if (hipblasSetStream(cublas_handle, stream[1]) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	if (hipblasSaxpy(cublas_handle, npix, &K11, d_zy, 1, d_N + npix, 1) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	hipStreamCreate(&stream[2]); CUDA_CHECK;
	third_and_fourth_normal_component << < (unsigned)(npix - 1) / 256 + 1, 256, 0, stream[2] >> > (d_z, d_xx, d_yy, d_zx, d_zy, K02, K12, npix, d_N + npix * 2); CUDA_CHECK;
	hipDeviceSynchronize(); CUDA_CHECK;
	norm_components << < (unsigned)(npix - 1) / 256 + 1, 256 >> > (d_N, npix, d_norm); CUDA_CHECK;
	hipDeviceSynchronize(); CUDA_CHECK;
	dim3 block(256, 1, 1);
	dim3 grid((unsigned)(npix - 1) / block.x + 1, 3, 1);
	normalize_N << < grid, block >> > (d_N, d_norm, npix); CUDA_CHECK;
	return d_N;
}

void cuda_based_preconditioned_conjugate_gradient(hipblasHandle_t& cublasHandle, hipsparseHandle_t& cusparseHandle, int* d_A_row, int* d_A_col, float* d_A_val, int N, int nnz, float* d_x, float* d_b) {
	// Will need to add COO sort
	const float tol = 1e-9f;
	const int max_iter = 100;
	float r1, alpha, beta;
	float rsum, diff, err = 0.0;
	float dot, numerator, denominator, nalpha;
	const float floatone = 1.0;
	const float floatzero = 0.0;
	hipsparseStatus_t cusparseStatus;
	hipsparseMatDescr_t descr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descr); CUSPARSE_CHECK(cusparseStatus);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
	float *d_zm1, *d_zm2, *d_rm2, *d_y, *d_p, *d_omega;
	float *d_valsILU0;
	int nzILU0 = 2 * N - 1; 
	hipMalloc((void **)&d_y, N * sizeof(float)); CUDA_CHECK;
	hipMalloc((void **)&d_p, N * sizeof(float)); CUDA_CHECK;
	hipMalloc((void **)&d_omega, N * sizeof(float)); CUDA_CHECK;
	hipMalloc((void **)&d_valsILU0, nnz * sizeof(float)); CUDA_CHECK;
	hipMalloc((void **)&d_zm1, (N) * sizeof(float)); CUDA_CHECK;
	hipMalloc((void **)&d_zm2, (N) * sizeof(float)); CUDA_CHECK;
	hipMalloc((void **)&d_rm2, (N) * sizeof(float)); CUDA_CHECK;
	/* create the analysis info object for the A matrix */
	cusparseSolveAnalysisInfo_t infoA = 0;
	cusparseStatus = cusparseCreateSolveAnalysisInfo(&infoA); CUSPARSE_CHECK(cusparseStatus);
	/* Perform the analysis for the Non-Transpose case */
	cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, descr, d_A_val, d_A_row, d_A_col, infoA); CUSPARSE_CHECK(cusparseStatus);
	/* Copy A data to ILU0 vals as input*/
	hipMemcpy(d_valsILU0, d_A_val, nnz * sizeof(float), hipMemcpyDeviceToDevice); CUDA_CHECK;
	/* generate the Incomplete LU factor H for the matrix A using cudsparseScsrilu0 */
	cusparseStatus = cusparseScsrilu0(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, descr, d_valsILU0, d_A_row, d_A_col, infoA); CUSPARSE_CHECK(cusparseStatus);
	/* Create info objects for the ILU0 preconditioner */
	cusparseSolveAnalysisInfo_t info_u;
	cusparseCreateSolveAnalysisInfo(&info_u);
	hipsparseMatDescr_t descrL = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descrL); CUSPARSE_CHECK(cusparseStatus);
	hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);
	hipsparseMatDescr_t descrU = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descrU); CUSPARSE_CHECK(cusparseStatus);
	hipsparseSetMatType(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrU, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
	hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, descrU, d_A_val, d_A_row, d_A_col, info_u); CUSPARSE_CHECK(cusparseStatus);
	int k = 0;
	hipblasSdot(cublasHandle, N, d_b, 1, d_b, 1, &r1);
	while (r1 > tol*tol && k <= max_iter) {
		// Forward Solve, we can re-use infoA since the sparsity pattern of A matches that of L
		cusparseStatus = cusparseScsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &floatone, descrL, d_valsILU0, d_A_row, d_A_col, infoA, d_b, d_y); CUSPARSE_CHECK(cusparseStatus);
		// Back Substitution
		cusparseStatus = cusparseScsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &floatone, descrU, d_valsILU0, d_A_row, d_A_col, info_u, d_y, d_zm1); CUSPARSE_CHECK(cusparseStatus);
		k++;
		if (k == 1) {
			hipblasScopy(cublasHandle, N, d_zm1, 1, d_p, 1);
		}
		else {
			hipblasSdot(cublasHandle, N, d_b, 1, d_zm1, 1, &numerator);
			hipblasSdot(cublasHandle, N, d_rm2, 1, d_zm2, 1, &denominator);
			beta = numerator / denominator;
			hipblasSscal(cublasHandle, N, &beta, d_p, 1);
			hipblasSaxpy(cublasHandle, N, &floatone, d_zm1, 1, d_p, 1);
		}
		hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nzILU0, &floatone, descrU, d_A_val, d_A_row, d_A_col, d_p, &floatzero, d_omega);
		hipblasSdot(cublasHandle, N, d_b, 1, d_zm1, 1, &numerator);
		hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &denominator);
		alpha = numerator / denominator;
		hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
		hipblasScopy(cublasHandle, N, d_b, 1, d_rm2, 1);
		hipblasScopy(cublasHandle, N, d_zm1, 1, d_zm2, 1);
		nalpha = -alpha;
		hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_b, 1);
		hipblasSdot(cublasHandle, N, d_b, 1, d_b, 1, &r1);
	}
	/* Destroy parameters */
	cusparseDestroySolveAnalysisInfo(infoA);
	cusparseDestroySolveAnalysisInfo(info_u);

	/* Free device memory */
	hipFree(d_y); CUDA_CHECK;
	hipFree(d_p); CUDA_CHECK;
	hipFree(d_omega); CUDA_CHECK;
	hipFree(d_valsILU0); CUDA_CHECK;
	hipFree(d_zm1); CUDA_CHECK;
	hipFree(d_zm2); CUDA_CHECK;
	hipFree(d_rm2); CUDA_CHECK;
}

__global__ void A_for_lightning_estimation(float* rho, float* N, int npix, float* A) {
	int i = blockIdx.x*blockDim.x + threadIdx.x; // pixel index
	int c = blockIdx.y*blockDim.y + threadIdx.y; // channel index
	int h = blockIdx.z*blockDim.z + threadIdx.z; // harmonic index
	if (i < npix) {
		A[c*npix*4+h*npix+i] = rho[c*npix + i] * N[h*npix + i];
	}
}

float* cuda_based_A_for_lightning(float* d_rho, float* d_N, int npix, int nchannels) {
	float* d_A;
	hipMalloc(&d_A, sizeof(float)*npix * 4 * nchannels);
	dim3 block(256, 1, 1);
	dim3 grid((unsigned)(npix - 1) / block.x + 1, (unsigned)(nchannels - 1) / block.y + 1, 4);
	A_for_lightning_estimation<<<grid, block>>>(d_rho, d_N, npix, d_A);
	hipDeviceSynchronize();
	return d_A;
}

void cuda_based_lightning_estimation(hipblasHandle_t cublas_handle, hipsparseHandle_t cusp_handle, float* d_s, float* d_rho, float* d_N, float* d_I, int npix, int nimages, int nchannels) {
	hipsparseStatus_t status;
	float* d_A = cuda_based_A_for_lightning(d_rho, d_N, npix, nchannels);
	WRITE_MAT_FROM_DEVICE(d_A, npix*4*nchannels, "A.mat");
	float* d_b = d_I;
	for (int i = 0; i < nimages; i++) {
		for (int j = 0; j < nchannels; j++){
			float* d_A_ij = d_A + j*npix * 4;
			float* d_b_ij = d_b + i*npix*nchannels + j*npix;
			float* d_x_ij = d_s + i * 4 * nchannels + j * 4;
			int *dANnzPerRow;
			float *dCsrValA, *dCsrValATA, *d_ATb_ij;
			int *dCsrRowPtrA, *dCsrColIndA, *dCsrRowPtrATA, *dCsrColIndATA;
			int totalANnz,nnzATA;
			hipsparseMatDescr_t Adescr = 0;
			hipMalloc((void **)&dANnzPerRow, sizeof(int) * npix); CUDA_CHECK;
			status = hipsparseCreateMatDescr(&Adescr); CUSPARSE_CHECK(status);
			hipsparseSetMatType(Adescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
			hipsparseSetMatIndexBase(Adescr, HIPSPARSE_INDEX_BASE_ZERO);
			status = hipsparseSnnz(cusp_handle, HIPSPARSE_DIRECTION_ROW, npix, 4, Adescr, d_A_ij, npix, dANnzPerRow, &totalANnz); CUSPARSE_CHECK(status);
			hipMalloc((void **)&dCsrValA, sizeof(float) * totalANnz); CUDA_CHECK;
			hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (npix + 1)); CUDA_CHECK;
			hipMalloc((void **)&dCsrColIndA, sizeof(int) * totalANnz); CUDA_CHECK;
			status = hipsparseSdense2csr(cusp_handle, npix, 4, Adescr, d_A_ij, npix, dANnzPerRow, dCsrValA, dCsrRowPtrA, dCsrColIndA); CUSPARSE_CHECK(status);
			hipFree(dANnzPerRow); CUDA_CHECK;
			hipMalloc((void**)&dCsrRowPtrATA, sizeof(int)*(4 + 1)); CUDA_CHECK;
			status = hipsparseXcsrgemmNnz(cusp_handle, HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 4, npix, Adescr, totalANnz, dCsrRowPtrA, dCsrColIndA, Adescr, totalANnz, dCsrRowPtrA, dCsrColIndA, Adescr, dCsrRowPtrATA, &nnzATA); CUSPARSE_CHECK(status);
			hipMalloc((void**)&dCsrColIndATA, sizeof(int)*nnzATA); CUDA_CHECK;
			hipMalloc((void**)&dCsrValATA, sizeof(float)*nnzATA); CUDA_CHECK;
			hipMalloc((void**)&d_ATb_ij, sizeof(float)*4); CUDA_CHECK;
			status = hipsparseScsrgemm(cusp_handle, HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 4, npix, Adescr, totalANnz, dCsrValA, dCsrRowPtrA, dCsrColIndA, Adescr, totalANnz, dCsrValA, dCsrRowPtrA, dCsrColIndA, Adescr, dCsrValATA, dCsrRowPtrATA, dCsrColIndATA); CUSPARSE_CHECK(status);
			float d_one = 1.f, d_zero = 0.f;
			status = hipsparseScsrmv(cusp_handle, HIPSPARSE_OPERATION_TRANSPOSE, npix, 4, totalANnz, &d_one, Adescr, dCsrValA, dCsrRowPtrA, dCsrColIndA, d_b_ij, &d_zero, d_ATb_ij); CUSPARSE_CHECK(status);
			hipFree(dCsrRowPtrA); CUDA_CHECK;
			hipFree(dCsrColIndA); CUDA_CHECK;
			hipFree(dCsrValA); CUDA_CHECK;
			status = hipsparseDestroyMatDescr(Adescr); CUSPARSE_CHECK(status);
			cuda_based_preconditioned_conjugate_gradient(cublas_handle, cusp_handle, dCsrRowPtrATA, dCsrColIndATA, dCsrValATA, 4, nnzATA, d_x_ij, d_ATb_ij);
			hipFree(dCsrValATA); CUDA_CHECK;
			hipFree(d_ATb_ij); CUDA_CHECK;
			hipFree(dCsrRowPtrATA); CUDA_CHECK;
			hipFree(dCsrColIndATA); CUDA_CHECK;
		}
	}
	WRITE_MAT_FROM_DEVICE(d_s, nimages*4*nchannels, "s.mat");
	hipFree(d_A);
}