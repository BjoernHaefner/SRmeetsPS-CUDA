#include "hip/hip_runtime.h"
#include <devicecalls.cuh>
#include "Exceptions.h"

float* sort_COO(hipsparseHandle_t cusp_handle, int n_rows, int n_cols, int nnz, int* d_row_ind, int* d_col_ind, float* d_vals_unsorted) {
	hipsparseStatus_t status;
	float* d_vals = NULL;
	size_t pBufferSizeInBytes = 0;
	void *pBuffer = NULL;
	int *P = NULL;
	hipMalloc(&d_vals, nnz * sizeof(float)); CUDA_CHECK;
	status = hipsparseXcoosort_bufferSizeExt(cusp_handle, n_rows, n_cols, nnz, d_row_ind, d_col_ind, &pBufferSizeInBytes); CUSPARSE_CHECK(status);
	hipMalloc(&pBuffer, sizeof(char)* pBufferSizeInBytes); CUDA_CHECK;
	hipMalloc((void**)&P, sizeof(int)*nnz); CUDA_CHECK;
	status = hipsparseCreateIdentityPermutation(cusp_handle, nnz, P); CUSPARSE_CHECK(status);
	status = hipsparseXcoosortByRow(cusp_handle, n_rows, n_cols, nnz, d_row_ind, d_col_ind, P, pBuffer); CUSPARSE_CHECK(status);
	status = hipsparseSgthr(cusp_handle, nnz, d_vals_unsorted, d_vals, P, HIPSPARSE_INDEX_BASE_ZERO); CUSPARSE_CHECK(status);
	hipFree(d_vals_unsorted); CUDA_CHECK;
	hipFree(P); CUDA_CHECK;
	hipFree(pBuffer); CUDA_CHECK;
	return d_vals;
}

float* cuda_based_sparsemat_densevec_mul(hipsparseHandle_t& cusp_handle, int* row_ind, int* col_ind, float* vals, int n_rows, int n_cols, int nnz, float* d_vector) {
	int* d_row_ind = NULL;
	int* d_row_csr = NULL;
	int* d_col_ind = NULL;
	float* d_vals = NULL;
	float* d_output = NULL;
	hipMalloc(&d_col_ind, nnz * sizeof(int)); CUDA_CHECK;
	hipMalloc(&d_row_ind, nnz * sizeof(int)); CUDA_CHECK;
	hipMalloc(&d_vals, nnz * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_output, n_rows * sizeof(float)); CUDA_CHECK;
	hipMemcpy(d_row_ind, row_ind, nnz * sizeof(int), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(d_col_ind, col_ind, nnz * sizeof(int), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(d_vals, vals, nnz * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	d_vals = sort_COO(cusp_handle, n_rows, n_cols, nnz, d_row_ind, d_col_ind, d_vals);
	hipMalloc(&d_row_csr, (n_rows + 1) * sizeof(int)); CUDA_CHECK;
	hipsparseStatus_t cusp_stat;
	hipsparseMatDescr_t cusp_mat_desc = 0;
	cusp_stat = hipsparseCreateMatDescr(&cusp_mat_desc);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Matrix descriptor initialization failed");
	}
	hipsparseSetMatType(cusp_mat_desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(cusp_mat_desc, HIPSPARSE_INDEX_BASE_ZERO);

	cusp_stat = hipsparseXcoo2csr(cusp_handle, d_row_ind, (int)nnz, (int)n_rows, d_row_csr, HIPSPARSE_INDEX_BASE_ZERO);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Conversion from COO to CSR format failed");
	}
	float d_one = 1.f, d_zero = 0.f;
	cusp_stat = hipsparseScsrmv(cusp_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, (int)n_rows, (int)n_cols, (int)nnz, &d_one, cusp_mat_desc, d_vals, d_row_csr, d_col_ind, d_vector, &d_zero, d_output);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Matrix-vector multiplication failed");
	}
	cusp_stat = hipsparseDestroyMatDescr(cusp_mat_desc);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Matrix descriptor destruction failed");
	}

	hipFree(d_row_ind); CUDA_CHECK;
	hipFree(d_row_csr); CUDA_CHECK;
	hipFree(d_col_ind); CUDA_CHECK;
	hipFree(d_vals); CUDA_CHECK;
	return d_output;
}

__global__ void mean_across_channels(float* data, int h, int w, int nc, float* mean, uint8_t* inpaint_locations) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < h && j < w) {
		float avg = 0.f;
		for (int c = 0; c < nc; c++) {
			if (data[c*(w*h) + j*h + i] != 0)
				avg += data[c*(w*h) + j*h + i];
			else {
				inpaint_locations[j*h + i] = 1;
				//avg = NAN;
			}
		}
		mean[j*h + i] = avg / nc;
	}
}

float* cuda_based_mean_across_channels(float* data, int h, int w, int nc, uint8_t** d_inpaint_locations) {
	float* d_data = NULL;
	float* d_output = NULL;
	dim3 block(128, 8, 1);
	dim3 grid((unsigned)(h - 1) / block.x + 1, (unsigned)(w - 1) / block.y + 1, 1);
	hipMalloc(&d_data, h * w * nc * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_output, h * w * sizeof(float)); CUDA_CHECK;
	hipMalloc(d_inpaint_locations, h * w * sizeof(uint8_t)); CUDA_CHECK;
	hipMemset(*d_inpaint_locations, 0, h * w * sizeof(uint8_t)); CUDA_CHECK;
	hipMemcpy(d_data, data, h * w * nc * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	mean_across_channels << <grid, block >> > (d_data, h, w, nc, d_output, *d_inpaint_locations); CUDA_CHECK;
	hipFree(d_data); CUDA_CHECK;
	return d_output;
}

float* cuda_based_image_resize(float* data, int h, int w, int new_h, int new_w) {
	// TODO: switch to cv cuda
	return NULL;
}


__global__ void initialize_rho(float* rho, int size_c, int nc) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int c = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < size_c && c < nc) {
		rho[c*(size_c)+i] = 0.5f;
	}
}

float* cuda_based_rho_init(thrust::host_vector<int>& imask, int nc) {
	float* d_rho = NULL;
	hipMalloc(&d_rho, imask.size() * nc * sizeof(float)); CUDA_CHECK;
	dim3 block(512, 1, 1);
	dim3 grid((unsigned)(imask.size() - 1) / block.x + 1, (unsigned)(nc - 1) / block.y + 1, 1);
	initialize_rho << < grid, block >> > (d_rho, (int)imask.size(), nc); CUDA_CHECK;
	hipDeviceSynchronize();
	return d_rho;
}

__global__ void meshgrid_create(float* xx, float* yy, int w, int h) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < h && j < w) {
		xx[j*h + i] = j;
		yy[j*h + i] = i;
	}
}

std::pair<float*, float*> cuda_based_meshgrid_create(int w, int h) {
	float* xx = NULL, *yy = NULL;
	hipMalloc(&xx, sizeof(float)*w*h); CUDA_CHECK;
	hipMalloc(&yy, sizeof(float)*w*h); CUDA_CHECK;
	dim3 block(32, 8, 1);
	dim3 grid((unsigned)(w - 1) / block.x + 1, (unsigned)(h - 1) / block.y + 1, 1);
	meshgrid_create << <grid, block >> > (xx, yy, w, h);
	hipDeviceSynchronize();
	return std::pair<float*, float*>(xx, yy);
}

__global__ void third_and_fourth_normal_component(float* z, float* xx, float* yy, float* zx, float* zy, float K02, float K12, int npix, float* N3) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < npix) {
		N3[i] = -z[i] - (xx[i] - K02) * zx[i] - (yy[i] - K12) * zy[i];
		N3[npix + i] = 1;
	}
}

__global__ void norm_components(float* N, int npix, float* norm) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < npix) {
		norm[i] = fmaxf(1e-10, sqrtf(N[i] * N[i] + N[npix + i] * N[npix + i] + N[npix * 2 + i] * N[npix * 2 + i]));
	}
}

__global__ void normalize_N(float* N, float* norm, int npix_per_component) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int c = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < npix_per_component) {
		N[c*npix_per_component + i] = N[c*npix_per_component + i] / norm[i];
	}
}

float* cuda_based_normal_init(hipblasHandle_t cublas_handle, float *d_z, float* d_zx, float *d_zy, float *d_xx, float *d_yy, int npix, float K00, float K11, float K02, float K12) {
	float* d_N = NULL;
	float* d_norm = NULL;
	hipStream_t stream[3];
	hipMalloc(&d_N, sizeof(float)*npix * 4); CUDA_CHECK;
	hipMemset(d_N, 0, sizeof(float)*npix * 4); CUDA_CHECK;
	hipMalloc(&d_norm, sizeof(float)*npix); CUDA_CHECK;
	hipStreamCreate(&stream[0]); CUDA_CHECK;
	if (hipblasSetStream(cublas_handle, stream[0]) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	if (hipblasSaxpy(cublas_handle, npix, &K00, d_zx, 1, d_N, 1) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	hipStreamCreate(&stream[1]); CUDA_CHECK;
	if (hipblasSetStream(cublas_handle, stream[1]) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	if (hipblasSaxpy(cublas_handle, npix, &K11, d_zy, 1, d_N + npix, 1) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	hipStreamCreate(&stream[2]); CUDA_CHECK;
	third_and_fourth_normal_component << < (unsigned)(npix - 1) / 256 + 1, 256, 0, stream[2] >> > (d_z, d_xx, d_yy, d_zx, d_zy, K02, K12, npix, d_N + npix * 2); CUDA_CHECK;
	hipDeviceSynchronize(); CUDA_CHECK;
	norm_components << < (unsigned)(npix - 1) / 256 + 1, 256 >> > (d_N, npix, d_norm); CUDA_CHECK;
	hipDeviceSynchronize(); CUDA_CHECK;
	dim3 block(256, 1, 1);
	dim3 grid((unsigned)(npix - 1) / block.x + 1, 3, 1);
	normalize_N << < grid, block >> > (d_N, d_norm, npix); CUDA_CHECK;
	return d_N;
}

void cuda_based_preconditioned_conjugate_gradient(hipblasHandle_t& cublasHandle, hipsparseHandle_t& cusparseHandle, int* d_A_row, int* d_A_col, float* d_A_val, int N, int nnz, float* d_x, float* d_b) {
	// Will need to add COO sort
	const float tol = 1e-9f;
	const int max_iter = 100;
	float r1, alpha, beta;
	float rsum, diff, err = 0.0;
	float dot, numerator, denominator, nalpha;
	const float floatone = 1.0;
	const float floatzero = 0.0;
	hipsparseStatus_t cusparseStatus;
	hipsparseMatDescr_t descr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descr); CUSPARSE_CHECK(cusparseStatus);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
	float *d_zm1, *d_zm2, *d_rm2, *d_y, *d_p, *d_omega;
	float *d_valsILU0;
	int nzILU0 = 2 * N - 1; 
	hipMalloc((void **)&d_y, N * sizeof(float)); CUDA_CHECK;
	hipMalloc((void **)&d_p, N * sizeof(float)); CUDA_CHECK;
	hipMalloc((void **)&d_omega, N * sizeof(float)); CUDA_CHECK;
	hipMalloc((void **)&d_valsILU0, nnz * sizeof(float)); CUDA_CHECK;
	hipMalloc((void **)&d_zm1, (N) * sizeof(float)); CUDA_CHECK;
	hipMalloc((void **)&d_zm2, (N) * sizeof(float)); CUDA_CHECK;
	hipMalloc((void **)&d_rm2, (N) * sizeof(float)); CUDA_CHECK;
	/* create the analysis info object for the A matrix */
	cusparseSolveAnalysisInfo_t infoA = 0;
	cusparseStatus = cusparseCreateSolveAnalysisInfo(&infoA); CUSPARSE_CHECK(cusparseStatus);
	/* Perform the analysis for the Non-Transpose case */
	cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, descr, d_A_val, d_A_row, d_A_col, infoA); CUSPARSE_CHECK(cusparseStatus);
	/* Copy A data to ILU0 vals as input*/
	hipMemcpy(d_valsILU0, d_A_val, nnz * sizeof(float), hipMemcpyDeviceToDevice); CUDA_CHECK;
	/* generate the Incomplete LU factor H for the matrix A using cudsparseScsrilu0 */
	cusparseStatus = cusparseScsrilu0(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, descr, d_valsILU0, d_A_row, d_A_col, infoA); CUSPARSE_CHECK(cusparseStatus);
	/* Create info objects for the ILU0 preconditioner */
	cusparseSolveAnalysisInfo_t info_u;
	cusparseCreateSolveAnalysisInfo(&info_u);
	hipsparseMatDescr_t descrL = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descrL); CUSPARSE_CHECK(cusparseStatus);
	hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);
	hipsparseMatDescr_t descrU = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descrU); CUSPARSE_CHECK(cusparseStatus);
	hipsparseSetMatType(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrU, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
	hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, descrU, d_A_val, d_A_row, d_A_col, info_u); CUSPARSE_CHECK(cusparseStatus);
	int k = 0;
	hipblasSdot(cublasHandle, N, d_b, 1, d_b, 1, &r1);
	while (r1 > tol*tol && k <= max_iter) {
		// Forward Solve, we can re-use infoA since the sparsity pattern of A matches that of L
		cusparseStatus = cusparseScsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &floatone, descrL, d_valsILU0, d_A_row, d_A_col, infoA, d_b, d_y); CUSPARSE_CHECK(cusparseStatus);
		// Back Substitution
		cusparseStatus = cusparseScsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &floatone, descrU, d_valsILU0, d_A_row, d_A_col, info_u, d_y, d_zm1); CUSPARSE_CHECK(cusparseStatus);
		k++;
		if (k == 1) {
			hipblasScopy(cublasHandle, N, d_zm1, 1, d_p, 1);
		}
		else {
			hipblasSdot(cublasHandle, N, d_b, 1, d_zm1, 1, &numerator);
			hipblasSdot(cublasHandle, N, d_rm2, 1, d_zm2, 1, &denominator);
			beta = numerator / denominator;
			hipblasSscal(cublasHandle, N, &beta, d_p, 1);
			hipblasSaxpy(cublasHandle, N, &floatone, d_zm1, 1, d_p, 1);
		}
		hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nzILU0, &floatone, descrU, d_A_val, d_A_row, d_A_col, d_p, &floatzero, d_omega);
		hipblasSdot(cublasHandle, N, d_b, 1, d_zm1, 1, &numerator);
		hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &denominator);
		alpha = numerator / denominator;
		hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
		hipblasScopy(cublasHandle, N, d_b, 1, d_rm2, 1);
		hipblasScopy(cublasHandle, N, d_zm1, 1, d_zm2, 1);
		nalpha = -alpha;
		hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_b, 1);
		hipblasSdot(cublasHandle, N, d_b, 1, d_b, 1, &r1);
	}
	/* Destroy parameters */
	cusparseDestroySolveAnalysisInfo(infoA);
	cusparseDestroySolveAnalysisInfo(info_u);

	/* Free device memory */
	hipFree(d_y); CUDA_CHECK;
	hipFree(d_p); CUDA_CHECK;
	hipFree(d_omega); CUDA_CHECK;
	hipFree(d_valsILU0); CUDA_CHECK;
	hipFree(d_zm1); CUDA_CHECK;
	hipFree(d_zm2); CUDA_CHECK;
	hipFree(d_rm2); CUDA_CHECK;
}

__global__ void A_for_lightning_estimation(float* rho, float* N, int npix, float* A) {
	int i = blockIdx.x*blockDim.x + threadIdx.x; // pixel index
	int c = blockIdx.y*blockDim.y + threadIdx.y; // channel index
	int h = blockIdx.z*blockDim.z + threadIdx.z; // harmonic index
	if (i < npix) {
		A[c*npix*4+h*npix+i] = rho[c*npix + i] * N[h*npix + i];
	}
}

float* cuda_based_A_for_lightning(float* d_rho, float* d_N, int npix, int nchannels) {
	float* d_A;
	hipMalloc(&d_A, sizeof(float)*npix * 4 * nchannels);
	dim3 block(256, 1, 1);
	dim3 grid((unsigned)(npix - 1) / block.x + 1, (unsigned)(nchannels - 1) / block.y + 1, 4);
	A_for_lightning_estimation<<<grid, block>>>(d_rho, d_N, npix, d_A);
	hipDeviceSynchronize();
	return d_A;
}

void cuda_based_ATA_ATb(hipsparseHandle_t cusp_handle, hipsparseMatDescr_t& descr_A, int* d_A_row_ptr, int* d_A_col_ind, float* d_A_val, float* d_b, float* d_x, int rows, int cols, int nnz, int** d_ATA_row_ptr, int** d_ATA_col_ind, float** d_ATA_val, int& nnz_ATA, float** d_ATb) {
	hipsparseStatus_t status;
	hipMalloc((void**)d_ATA_row_ptr, sizeof(int)*(cols + 1)); CUDA_CHECK;
	status = hipsparseXcsrgemmNnz(cusp_handle, HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, cols, cols, rows, descr_A, nnz, d_A_row_ptr, d_A_col_ind, descr_A, nnz, d_A_row_ptr, d_A_col_ind, descr_A, *d_ATA_row_ptr, &nnz_ATA); CUSPARSE_CHECK(status);
	hipMalloc((void**)d_ATA_col_ind, sizeof(int)*nnz_ATA); CUDA_CHECK;
	hipMalloc((void**)d_ATA_val, sizeof(float)*nnz_ATA); CUDA_CHECK;
	status = hipsparseScsrgemm(cusp_handle, HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, cols, cols, rows, descr_A, nnz, d_A_val, d_A_row_ptr, d_A_col_ind, descr_A, nnz, d_A_val, d_A_row_ptr, d_A_col_ind, descr_A, *d_ATA_val, *d_ATA_row_ptr, *d_ATA_col_ind); CUSPARSE_CHECK(status);
	float d_one = 1.f, d_zero = 0.f, d_neg_one= -1.f;
	hipMalloc((void**)d_ATb, sizeof(float) * cols); CUDA_CHECK;
	status = hipsparseScsrmv(cusp_handle, HIPSPARSE_OPERATION_TRANSPOSE, rows, cols, nnz, &d_one, descr_A, d_A_val, d_A_row_ptr, d_A_col_ind, d_b, &d_zero, *d_ATb); CUSPARSE_CHECK(status);
	status = hipsparseScsrmv(cusp_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, cols, cols, nnz, &d_neg_one, descr_A, *d_ATA_val, *d_ATA_row_ptr, *d_ATA_col_ind, d_x, &d_one, *d_ATb); CUSPARSE_CHECK(status);
}

void cuda_based_lightning_estimation(hipblasHandle_t cublas_handle, hipsparseHandle_t cusp_handle, float* d_s, float* d_rho, float* d_N, float* d_I, int npix, int nimages, int nchannels) {
	hipsparseStatus_t status;
	float* d_A = cuda_based_A_for_lightning(d_rho, d_N, npix, nchannels);
	WRITE_MAT_FROM_DEVICE(d_A, npix*4*nchannels, "A.mat");
	float* d_b = d_I;
	for (int i = 0; i < nimages; i++) {
		for (int j = 0; j < nchannels; j++){
			float* d_A_ij = d_A + j*npix * 4;
			float* d_b_ij = d_b + i*npix*nchannels + j*npix;
			float* d_x_ij = d_s + i * 4 * nchannels + j * 4;

			int *dANnzPerRow = NULL;
			float *dCsrValA = NULL, *dCsrValATA = NULL, *d_ATb_ij = NULL;
			int *dCsrRowPtrA = NULL, *dCsrColIndA = NULL, *dCsrRowPtrATA = NULL, *dCsrColIndATA = NULL;
			int totalANnz, nnzATA = 0;
			hipsparseMatDescr_t Adescr = 0;
			hipMalloc((void **)&dANnzPerRow, sizeof(int) * npix); CUDA_CHECK;
			status = hipsparseCreateMatDescr(&Adescr); CUSPARSE_CHECK(status);
			hipsparseSetMatType(Adescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
			hipsparseSetMatIndexBase(Adescr, HIPSPARSE_INDEX_BASE_ZERO);
			status = hipsparseSnnz(cusp_handle, HIPSPARSE_DIRECTION_ROW, npix, 4, Adescr, d_A_ij, npix, dANnzPerRow, &totalANnz); CUSPARSE_CHECK(status);
			hipMalloc((void **)&dCsrValA, sizeof(float) * totalANnz); CUDA_CHECK;
			hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (npix + 1)); CUDA_CHECK;
			hipMalloc((void **)&dCsrColIndA, sizeof(int) * totalANnz); CUDA_CHECK;
			status = hipsparseSdense2csr(cusp_handle, npix, 4, Adescr, d_A_ij, npix, dANnzPerRow, dCsrValA, dCsrRowPtrA, dCsrColIndA); CUSPARSE_CHECK(status);
			hipFree(dANnzPerRow); CUDA_CHECK;
			cuda_based_ATA_ATb(cusp_handle, Adescr, dCsrRowPtrA, dCsrColIndA, dCsrValA, d_b_ij, d_x_ij, npix, 4, totalANnz, &dCsrRowPtrATA, &dCsrColIndATA, &dCsrValATA, nnzATA, &d_ATb_ij);
			hipFree(dCsrRowPtrA); CUDA_CHECK;
			hipFree(dCsrColIndA); CUDA_CHECK;
			hipFree(dCsrValA); CUDA_CHECK;
			
			status = hipsparseDestroyMatDescr(Adescr); CUSPARSE_CHECK(status);
			
			cuda_based_preconditioned_conjugate_gradient(cublas_handle, cusp_handle, dCsrRowPtrATA, dCsrColIndATA, dCsrValATA, 4, nnzATA, d_x_ij, d_ATb_ij);
			WRITE_MAT_FROM_DEVICE(d_x_ij, 4, "x_solved.mat");
			hipFree(dCsrValATA); CUDA_CHECK;
			hipFree(d_ATb_ij); CUDA_CHECK;
			hipFree(dCsrRowPtrATA); CUDA_CHECK;
			hipFree(dCsrColIndATA); CUDA_CHECK;
		}
	}
	WRITE_MAT_FROM_DEVICE(d_s, nimages*4*nchannels, "s.mat");
	hipFree(d_A);
}


__global__ void fill_A_albedo_COO(float* A, int* rowind, int* colind, float* val, int npix, int nimages) {
	int i = blockIdx.x*blockDim.x + threadIdx.x; 
	if (i < npix*nimages) {
		rowind[i] = i;
		colind[i] = i % npix;
		val[i] = A[i];
	}
}

void cuda_based_A_for_albedo(hipblasHandle_t cublas_handle, hipsparseHandle_t cusp_handle, float* d_N, float* d_s, int npix, int nchannels, int nimages, int** d_rowptr, int** d_colind, float** d_val){
	hipblasStatus_t status_cb;
	hipsparseStatus_t status_cs;
	hipsparseMatDescr_t cusp_mat_desc = 0;
	float* d_A, *d_s_buff;
	float d_one = 1.f, d_zero = 0.f;
	hipMalloc(&d_s_buff, sizeof(float) * 4 * nimages); CUDA_CHECK;
	for (int i = 0; i < nimages; i++) {
		hipMemcpy(d_s_buff + i * 4, d_s + i * 4 * nchannels, 4 * sizeof(float), hipMemcpyDeviceToDevice); CUDA_CHECK;
	}
	hipMalloc(&d_A, sizeof(float) * npix * nimages); CUDA_CHECK;
	status_cb = hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, npix, nimages, 4, &d_one, d_N, npix, d_s_buff, 4, &d_zero, d_A, npix); CUBLAS_CHECK(status_cb);
	int* d_rowind;
	hipMalloc(&d_rowind, npix * nimages * sizeof(int)); CUDA_CHECK;
	hipMalloc(d_colind, npix * nimages * sizeof(int)); CUDA_CHECK;
	hipMalloc(d_val, npix * nimages * sizeof(float)); CUDA_CHECK;
	fill_A_albedo_COO <<<(unsigned)(npix*nimages - 1) / 512 + 1, 512 >> > (d_A, d_rowind, *d_colind, *d_val, npix, nimages); CUDA_CHECK;
	hipMalloc(d_rowptr, (npix*nimages + 1) * sizeof(int)); CUDA_CHECK;
	hipsparseCreateMatDescr(&cusp_mat_desc);
	hipsparseSetMatType(cusp_mat_desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(cusp_mat_desc, HIPSPARSE_INDEX_BASE_ZERO);
	status_cs = hipsparseXcoo2csr(cusp_handle, d_rowind, npix*nimages, npix*nimages, *d_rowptr, HIPSPARSE_INDEX_BASE_ZERO); CUSPARSE_CHECK(status_cs);
	status_cs = hipsparseDestroyMatDescr(cusp_mat_desc); CUSPARSE_CHECK(status_cs);
	hipFree(d_A); CUDA_CHECK;
	hipFree(d_rowind); CUDA_CHECK;
	hipFree(d_s_buff); CUDA_CHECK;
}



void cuda_based_albedo_estimation(hipblasHandle_t cublas_handle, hipsparseHandle_t cusp_handle, float* d_s, float* d_rho, float* d_N, float* d_I, int npix, int nimages, int nchannels) {
	hipsparseMatDescr_t descr_A = 0;
	hipsparseCreateMatDescr(&descr_A);
	hipsparseSetMatType(descr_A, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr_A, HIPSPARSE_INDEX_BASE_ZERO);
	for (int c = 0; c < nchannels; c++){
		int* d_A_row_ptr = NULL, *d_A_col_ind = NULL;
		float* d_A_val = NULL;
		cuda_based_A_for_albedo(cublas_handle, cusp_handle, d_N, d_s + c * 4, npix, nchannels, nimages, &d_A_row_ptr, &d_A_col_ind, &d_A_val);
		float* d_b = NULL;
		hipMalloc(&d_b, npix*nimages * sizeof(float)); CUDA_CHECK;
		for (int i = 0; i < nimages; i++) {
			hipMemcpy(d_b + npix*i, d_I + c*npix + i*npix*nchannels, npix * sizeof(float), hipMemcpyDeviceToDevice); CUDA_CHECK;
		}
		float *d_ATA_val = NULL, *d_ATb = NULL;
		int* d_ATA_row_ptr = NULL, *d_ATA_col_ind = NULL;
		int nnz_ATA;
		cuda_based_ATA_ATb(cusp_handle, descr_A, d_A_row_ptr, d_A_col_ind, d_A_val, d_b, d_rho + npix*c, npix*nimages, npix, npix*nimages, &d_ATA_row_ptr, &d_ATA_col_ind, &d_ATA_val, nnz_ATA, &d_ATb);
		hipFree(d_A_row_ptr); CUDA_CHECK;
		hipFree(d_A_col_ind); CUDA_CHECK;
		hipFree(d_A_val); CUDA_CHECK;
		hipFree(d_b); CUDA_CHECK;
		cuda_based_preconditioned_conjugate_gradient(cublas_handle, cusp_handle, d_ATA_row_ptr, d_ATA_col_ind, d_ATA_val, npix, nnz_ATA, d_rho + npix*c, d_ATb);
		hipFree(d_ATA_val);
		hipFree(d_ATb);
		hipFree(d_ATA_row_ptr);
		hipFree(d_ATA_col_ind);
	}
	hipsparseDestroyMatDescr(descr_A);

}