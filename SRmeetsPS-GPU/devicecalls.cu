#include "hip/hip_runtime.h"
#include <devicecalls.cuh>
#include "Exceptions.h"

float* sort_COO(hipsparseHandle_t cusp_handle, int n_rows, int n_cols, int nnz, int* d_row_ind, int* d_col_ind, float* d_vals_unsorted) {
	float* d_vals = NULL;
	size_t pBufferSizeInBytes = 0;
	void *pBuffer = NULL;
	int *P = NULL;
	hipMalloc(&d_vals, nnz * sizeof(float)); CUDA_CHECK;
	hipsparseXcoosort_bufferSizeExt(cusp_handle, n_rows, n_cols, nnz, d_row_ind, d_col_ind, &pBufferSizeInBytes);
	hipMalloc(&pBuffer, sizeof(char)* pBufferSizeInBytes);
	hipMalloc((void**)&P, sizeof(int)*nnz);
	hipsparseCreateIdentityPermutation(cusp_handle, nnz, P);
	hipsparseXcoosortByRow(cusp_handle, n_rows, n_cols, nnz, d_row_ind, d_col_ind, P, pBuffer);
	hipsparseSgthr(cusp_handle, nnz, d_vals_unsorted, d_vals, P, HIPSPARSE_INDEX_BASE_ZERO);
	hipFree(d_vals_unsorted);
	hipFree(P);
	hipFree(pBuffer);
	return d_vals;
}

float* cuda_based_sparsemat_densevec_mul(hipsparseHandle_t& cusp_handle, int* row_ind, int* col_ind, float* vals, int n_rows, int n_cols, int nnz, float* d_vector) {
	int* d_row_ind = NULL;
	int* d_row_csr = NULL;
	int* d_col_ind = NULL;
	float* d_vals = NULL;
	float* d_output = NULL;
	hipMalloc(&d_col_ind, nnz * sizeof(int)); CUDA_CHECK;
	hipMalloc(&d_row_ind, nnz * sizeof(int)); CUDA_CHECK;
	hipMalloc(&d_vals, nnz * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_output, n_rows * sizeof(float)); CUDA_CHECK;
	hipMemcpy(d_row_ind, row_ind, nnz * sizeof(int), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(d_col_ind, col_ind, nnz * sizeof(int), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(d_vals, vals, nnz * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	d_vals = sort_COO(cusp_handle, n_rows, n_cols, nnz, d_row_ind, d_col_ind, d_vals);
	hipMalloc(&d_row_csr, (n_rows + 1) * sizeof(int)); CUDA_CHECK;
	hipsparseStatus_t cusp_stat;
	hipsparseMatDescr_t cusp_mat_desc = 0;
	cusp_stat = hipsparseCreateMatDescr(&cusp_mat_desc);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Matrix descriptor initialization failed");
	}
	hipsparseSetMatType(cusp_mat_desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(cusp_mat_desc, HIPSPARSE_INDEX_BASE_ZERO);

	cusp_stat = hipsparseXcoo2csr(cusp_handle, d_row_ind, (int)nnz, (int)n_rows, d_row_csr, HIPSPARSE_INDEX_BASE_ZERO);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Conversion from COO to CSR format failed");
	}
	float d_one = 1.f, d_zero = 0.f;
	cusp_stat = hipsparseScsrmv(cusp_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, (int)n_rows, (int)n_cols, (int)nnz, &d_one, cusp_mat_desc, d_vals, d_row_csr, d_col_ind, d_vector, &d_zero, d_output);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Matrix-vector multiplication failed");
	}
	cusp_stat = hipsparseDestroyMatDescr(cusp_mat_desc);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Matrix descriptor destruction failed");
	}

	hipFree(d_row_ind); CUDA_CHECK;
	hipFree(d_row_csr); CUDA_CHECK;
	hipFree(d_col_ind); CUDA_CHECK;
	hipFree(d_vals); CUDA_CHECK;
	return d_output;
}

__global__ void mean_across_channels(float* data, int h, int w, int nc, float* mean, uint8_t* inpaint_locations) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < h && j < w) {
		float avg = 0.f;
		for (int c = 0; c < nc; c++) {
			if (data[c*(w*h) + j*h + i] != 0)
				avg += data[c*(w*h) + j*h + i];
			else {
				inpaint_locations[j*h + i] = 1;
				//avg = NAN;
			}
		}
		mean[j*h + i] = avg / nc;
	}
}

float* cuda_based_mean_across_channels(float* data, int h, int w, int nc, uint8_t** d_inpaint_locations) {
	float* d_data = NULL;
	float* d_output = NULL;
	dim3 block(128, 8, 1);
	dim3 grid((unsigned)(h - 1) / block.x + 1, (unsigned)(w - 1) / block.y + 1, 1);
	hipMalloc(&d_data, h * w * nc * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_output, h * w * sizeof(float)); CUDA_CHECK;
	hipMalloc(d_inpaint_locations, h * w * sizeof(uint8_t)); CUDA_CHECK;
	hipMemset(*d_inpaint_locations, 0, h * w * sizeof(uint8_t)); CUDA_CHECK;
	hipMemcpy(d_data, data, h * w * nc * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	mean_across_channels << <grid, block >> > (d_data, h, w, nc, d_output, *d_inpaint_locations); CUDA_CHECK;
	hipFree(d_data); CUDA_CHECK;
	return d_output;
}

float* cuda_based_image_resize(float* data, int h, int w, int new_h, int new_w) {
	// TODO: switch to cv cuda
	return NULL;
}


__global__ void initialize_rho(float* rho, int size_c, int nc) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int c = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < size_c && c < nc) {
		rho[c*(size_c)+i] = 0.5f;
	}
}

float* cuda_based_rho_init(thrust::host_vector<int>& imask, int nc) {
	float* d_rho = NULL;
	hipMalloc(&d_rho, imask.size() * nc * sizeof(float)); CUDA_CHECK;
	dim3 block(512, 1, 1);
	dim3 grid((unsigned)(imask.size() - 1) / block.x + 1, (unsigned)(nc - 1) / block.y + 1, 1);
	initialize_rho << < grid, block >> > (d_rho, (int)imask.size(), nc); CUDA_CHECK;
	hipDeviceSynchronize();
	return d_rho;
}

__global__ void meshgrid_create(float* xx, float* yy, int w, int h) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < h && j < w) {
		xx[j*h + i] = j;
		yy[j*h + i] = i;
	}
}

std::pair<float*, float*> cuda_based_meshgrid_create(int w, int h) {
	float* xx = NULL, *yy = NULL;
	hipMalloc(&xx, sizeof(float)*w*h); CUDA_CHECK;
	hipMalloc(&yy, sizeof(float)*w*h); CUDA_CHECK;
	dim3 block(32, 8, 1);
	dim3 grid((unsigned)(w - 1) / block.x + 1, (unsigned)(h - 1) / block.y + 1, 1);
	meshgrid_create << <grid, block >> > (xx, yy, w, h);
	hipDeviceSynchronize();
	return std::pair<float*, float*>(xx, yy);
}

__global__ void third_and_fourth_normal_component(float* z, float* xx, float* yy, float* zx, float* zy, float K02, float K12, int npix, float* N3) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < npix) {
		N3[i] = -z[i] - (xx[i] - K02) * zx[i] - (yy[i] - K12) * zy[i];
		N3[npix + i] = 1;
	}
}

__global__ void norm_components(float* N, int npix, float* norm) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < npix) {
		norm[i] = fmaxf(1e-10, sqrtf(N[i] * N[i] + N[npix + i] * N[npix + i] + N[npix * 2 + i] * N[npix * 2 + i]));
	}
}

__global__ void normalize_N(float* N, float* norm, int npix_per_component) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int c = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < npix_per_component) {
		N[c*npix_per_component + i] = N[c*npix_per_component + i] / norm[i];
	}
}

float* cuda_based_normal_init(hipblasHandle_t cublas_handle, float *d_z, float* d_zx, float *d_zy, float *d_xx, float *d_yy, int npix, float K00, float K11, float K02, float K12) {
	float* d_N = NULL;
	float* d_norm = NULL;
	hipStream_t stream[3];
	hipMalloc(&d_N, sizeof(float)*npix * 4); CUDA_CHECK;
	hipMemset(d_N, 0, sizeof(float)*npix * 4); CUDA_CHECK;
	hipMalloc(&d_norm, sizeof(float)*npix); CUDA_CHECK;
	hipStreamCreate(&stream[0]); CUDA_CHECK;
	if (hipblasSetStream(cublas_handle, stream[0]) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	if (hipblasSaxpy(cublas_handle, npix, &K00, d_zx, 1, d_N, 1) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	hipStreamCreate(&stream[1]); CUDA_CHECK;
	if (hipblasSetStream(cublas_handle, stream[1]) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	if (hipblasSaxpy(cublas_handle, npix, &K11, d_zy, 1, d_N + npix, 1) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	hipStreamCreate(&stream[2]); CUDA_CHECK;
	third_and_fourth_normal_component << < (unsigned)(npix - 1) / 256 + 1, 256, 0, stream[2] >> > (d_z, d_xx, d_yy, d_zx, d_zy, K02, K12, npix, d_N + npix * 2); CUDA_CHECK;
	hipDeviceSynchronize(); CUDA_CHECK;
	norm_components << < (unsigned)(npix - 1) / 256 + 1, 256 >> > (d_N, npix, d_norm); CUDA_CHECK;
	hipDeviceSynchronize(); CUDA_CHECK;
	dim3 block(256, 1, 1);
	dim3 grid((unsigned)(npix - 1) / block.x + 1, 3, 1);
	normalize_N << < grid, block >> > (d_N, d_norm, npix); CUDA_CHECK;
	return d_N;
}

void cuda_based_preconditioned_conjugate_gradient(hipblasHandle_t& cublasHandle, hipsparseHandle_t& cusparseHandle, int* d_A_row, int* d_A_col, float* d_A_val, float* d_x, float* d_b, int N, int nnz) {
	// Will need to add COO sort
	const float tol = 1e-12f;
	const int max_iter = 1000;
	float r1, alpha, beta;
	float rsum, diff, err = 0.0;
	float dot, numerator, denominator, nalpha;
	const float floatone = 1.0;
	const float floatzero = 0.0;
	hipblasStatus_t hipblasStatus_t;
	hipsparseStatus_t cusparseStatus;
	hipsparseMatDescr_t descr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
	float *d_zm1, *d_zm2, *d_rm2, *d_y, *d_p, *d_omega;
	float *d_valsILU0;
	int nzILU0 = 2 * N - 1; 
	hipMalloc((void **)&d_y, N * sizeof(float));
	hipMalloc((void **)&d_p, N * sizeof(float));
	hipMalloc((void **)&d_omega, N * sizeof(float));
	hipMalloc((void **)&d_valsILU0, nnz * sizeof(float));
	hipMalloc((void **)&d_zm1, (N) * sizeof(float));
	hipMalloc((void **)&d_zm2, (N) * sizeof(float));
	hipMalloc((void **)&d_rm2, (N) * sizeof(float));
	/* create the analysis info object for the A matrix */
	cusparseSolveAnalysisInfo_t infoA = 0;
	cusparseStatus = cusparseCreateSolveAnalysisInfo(&infoA);
	/* Perform the analysis for the Non-Transpose case */
	cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
		N, nnz, descr, d_A_val, d_A_row, d_A_col, infoA);
	/* Copy A data to ILU0 vals as input*/
	hipMemcpy(d_valsILU0, d_A_val, nnz * sizeof(float), hipMemcpyDeviceToDevice);
	/* generate the Incomplete LU factor H for the matrix A using cudsparseScsrilu0 */
	cusparseStatus = cusparseScsrilu0(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, descr, d_valsILU0, d_A_row, d_A_col, infoA);
	/* Create info objects for the ILU0 preconditioner */
	cusparseSolveAnalysisInfo_t info_u;
	cusparseCreateSolveAnalysisInfo(&info_u);
	hipsparseMatDescr_t descrL = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descrL);
	hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);
	hipsparseMatDescr_t descrU = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descrU);
	hipsparseSetMatType(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrU, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
	hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, descrU, d_A_val, d_A_row, d_A_col, info_u);
	int k = 0;
	hipblasSdot(cublasHandle, N, d_b, 1, d_b, 1, &r1);
	while (r1 > tol*tol && k <= max_iter) {
		// Forward Solve, we can re-use infoA since the sparsity pattern of A matches that of L
		cusparseStatus = cusparseScsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &floatone, descrL,
			d_valsILU0, d_A_row, d_A_col, infoA, d_b, d_y);
		// Back Substitution
		cusparseStatus = cusparseScsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &floatone, descrU,
			d_valsILU0, d_A_row, d_A_col, info_u, d_y, d_zm1);
		k++;

		if (k == 1)
		{
			hipblasScopy(cublasHandle, N, d_zm1, 1, d_p, 1);
		}
		else
		{
			hipblasSdot(cublasHandle, N, d_b, 1, d_zm1, 1, &numerator);
			hipblasSdot(cublasHandle, N, d_rm2, 1, d_zm2, 1, &denominator);
			beta = numerator / denominator;
			hipblasSscal(cublasHandle, N, &beta, d_p, 1);
			hipblasSaxpy(cublasHandle, N, &floatone, d_zm1, 1, d_p, 1);
		}
		hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nzILU0, &floatone, descrU, d_A_val, d_A_row, d_A_col, d_p, &floatzero, d_omega);
		hipblasSdot(cublasHandle, N, d_b, 1, d_zm1, 1, &numerator);
		hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &denominator);
		alpha = numerator / denominator;
		hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
		hipblasScopy(cublasHandle, N, d_b, 1, d_rm2, 1);
		hipblasScopy(cublasHandle, N, d_zm1, 1, d_zm2, 1);
		nalpha = -alpha;
		hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_b, 1);
		hipblasSdot(cublasHandle, N, d_b, 1, d_b, 1, &r1);
	}
	/* Destroy parameters */
	cusparseDestroySolveAnalysisInfo(infoA);
	cusparseDestroySolveAnalysisInfo(info_u);

	/* Free device memory */
	hipFree(d_y);
	hipFree(d_p);
	hipFree(d_omega);
	hipFree(d_valsILU0);
	hipFree(d_zm1);
	hipFree(d_zm2);
	hipFree(d_rm2);
}