#include "hip/hip_runtime.h"
#include <devicecalls.cuh>
#include "Exceptions.h"

float* cuda_based_sparsemat_densevec_mul(hipsparseHandle_t& cusp_handle, int* row_ind, int* col_ind, float* vals, int n_rows, int n_cols, int nnz, float* d_vector) {
	int* d_row_ind = NULL;
	int* d_row_csr = NULL;
	int* d_col_ind = NULL;
	float* d_vals = NULL;
	float* d_output = NULL;
	hipMalloc(&d_col_ind, nnz * sizeof(int)); CUDA_CHECK;
	hipMalloc(&d_row_ind, nnz * sizeof(int)); CUDA_CHECK;
	hipMalloc(&d_vals, nnz * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_output, n_rows * sizeof(float)); CUDA_CHECK;
	hipMemcpy(d_row_ind, row_ind, nnz * sizeof(int), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(d_col_ind, col_ind, nnz * sizeof(int), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(d_vals, vals, nnz * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMalloc(&d_row_csr, (n_rows + 1) * sizeof(int)); CUDA_CHECK;

	hipsparseStatus_t cusp_stat;
	hipsparseMatDescr_t cusp_mat_desc = 0;

	cusp_stat = hipsparseCreateMatDescr(&cusp_mat_desc);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Matrix descriptor initialization failed");
	}
	hipsparseSetMatType(cusp_mat_desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(cusp_mat_desc, HIPSPARSE_INDEX_BASE_ZERO);

	cusp_stat = hipsparseXcoo2csr(cusp_handle, d_row_ind, (int)nnz, (int)n_rows, d_row_csr, HIPSPARSE_INDEX_BASE_ZERO);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Conversion from COO to CSR format failed");
	}
	float d_one = 1.f, d_zero = 0.f;
	cusp_stat = hipsparseScsrmv(cusp_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, (int)n_rows, (int)n_cols, (int)nnz, &d_one, cusp_mat_desc, d_vals, d_row_csr, d_col_ind, d_vector, &d_zero, d_output);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Matrix-vector multiplication failed");
	}
	cusp_stat = hipsparseDestroyMatDescr(cusp_mat_desc);
	if (cusp_stat != HIPSPARSE_STATUS_SUCCESS) {
		throw std::runtime_error("Matrix descriptor destruction failed");
	}

	hipFree(d_row_ind); CUDA_CHECK;
	hipFree(d_row_csr); CUDA_CHECK;
	hipFree(d_col_ind); CUDA_CHECK;
	hipFree(d_vals); CUDA_CHECK;
	return d_output;
}

__global__ void mean_across_channels(float* data, int h, int w, int nc, float* mean, uint8_t* inpaint_locations) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < h && j < w) {
		float avg = 0.f;
		for (int c = 0; c < nc; c++) {
			if (data[c*(w*h) + j*h + i] != 0)
				avg += data[c*(w*h) + j*h + i];
			else {
				inpaint_locations[j*h + i] = 1;
				//avg = NAN;
			}
		}
		mean[j*h + i] = avg / nc;
	}
}

float* cuda_based_mean_across_channels(float* data, int h, int w, int nc, uint8_t** d_inpaint_locations) {
	float* d_data = NULL;
	float* d_output = NULL;
	dim3 block(128, 8, 1);
	dim3 grid((unsigned)(h - 1) / block.x + 1, (unsigned)(w - 1) / block.y + 1, 1);
	hipMalloc(&d_data, h * w * nc * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_output, h * w * sizeof(float)); CUDA_CHECK;
	hipMalloc(d_inpaint_locations, h * w * sizeof(uint8_t)); CUDA_CHECK;
	hipMemset(*d_inpaint_locations, 0, h * w * sizeof(uint8_t)); CUDA_CHECK;
	hipMemcpy(d_data, data, h * w * nc * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	mean_across_channels << <grid, block >> > (d_data, h, w, nc, d_output, *d_inpaint_locations); CUDA_CHECK;
	hipFree(d_data); CUDA_CHECK;
	return d_output;
}

float* cuda_based_image_resize(float* data, int h, int w, int new_h, int new_w) {
	// TODO: switch to cv cuda
	return NULL;
}


__global__ void initialize_rho(float* rho, int size_c, int nc) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int c = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < size_c && c < nc) {
		rho[c*(size_c)+i] = 0.5f;
	}
}

float* cuda_based_rho_init(thrust::host_vector<int>& imask, int nc) {
	float* d_rho = NULL;
	hipMalloc(&d_rho, imask.size() * nc * sizeof(float)); CUDA_CHECK;
	dim3 block(512, 1, 1);
	dim3 grid((unsigned)(imask.size() - 1) / block.x + 1, (unsigned)(nc - 1) / block.y + 1, 1);
	initialize_rho <<< grid, block >>> (d_rho, (int)imask.size(), nc); CUDA_CHECK;
	hipDeviceSynchronize();
	return d_rho;
}

__global__ void meshgrid_create(float* xx, float* yy, int w, int h) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < h && j < w) {
		xx[j*h + i] = j;
		yy[j*h + i] = i;
	}
}

std::pair<float*, float*> cuda_based_meshgrid_create(int w, int h) {
	float* xx = NULL, *yy = NULL;
	hipMalloc(&xx, sizeof(float)*w*h); CUDA_CHECK;
	hipMalloc(&yy, sizeof(float)*w*h); CUDA_CHECK;
	dim3 block(32, 8, 1);
	dim3 grid((unsigned)(w - 1) / block.x + 1, (unsigned)(h - 1) / block.y + 1, 1);
	meshgrid_create << <grid, block >> > (xx, yy, w, h);
	hipDeviceSynchronize();
	return std::pair<float*, float*>(xx, yy);
}

__global__ void third_and_fourth_normal_component(float* z, float* xx, float* yy, float* zx, float* zy, float K02, float K12, int npix, float* N3) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < npix) {
		N3[i] = -z[i] - (xx[i]-K02) * zx[i] - (yy[i]-K12) * zy[i];
		N3[npix + i] = 1;
	}
}

__global__ void norm_components(float* N, int npix, float* norm) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < npix) {
		norm[i] = fmaxf(1e-10,sqrtf(N[i]* N[i] + N[npix + i]* N[npix + i] + N[npix * 2 + i]* N[npix * 2 + i]));
	}
}

__global__ void normalize_N(float* N, float* norm, int npix_per_component) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int c = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < npix_per_component) {
		N[c*npix_per_component + i] = N[c*npix_per_component + i] / norm[i];
	}
}

float* cuda_based_normal_init(hipblasHandle_t cublas_handle, float *d_z, float* d_zx, float *d_zy, float *d_xx, float *d_yy, int h, int w, float K00, float K11, float K02, float K12) {
	float* d_N = NULL;
	float* d_norm = NULL;
	hipStream_t stream[3];
	hipMalloc(&d_N, sizeof(float)*w*h * 4); CUDA_CHECK;
	hipMemset(d_N, 0, sizeof(float)*w*h * 4); CUDA_CHECK;
	hipMalloc(&d_norm, sizeof(float)*w*h); CUDA_CHECK;
	hipStreamCreate(&stream[0]); CUDA_CHECK;
	if (hipblasSetStream(cublas_handle, stream[0]) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	if (hipblasSaxpy(cublas_handle, w*h, &K00, d_zx, 1, d_N, 1) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	} 
	hipStreamCreate(&stream[1]); CUDA_CHECK;
	if (hipblasSetStream(cublas_handle, stream[1]) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	if (hipblasSaxpy(cublas_handle, w*h, &K11, d_zy, 1, d_N + w*h, 1) != HIPBLAS_STATUS_SUCCESS) {
		throw std::runtime_error("CUBLAS Library release of resources failed");
	}
	hipStreamCreate(&stream[2]); CUDA_CHECK;
	third_and_fourth_normal_component <<< (unsigned)(w*h - 1) / 256 + 1, 256, 0, stream[2] >>>(d_z, d_xx, d_yy, d_zx, d_zy, K02, K12, w*h, d_N+w*h*2); CUDA_CHECK;
	hipDeviceSynchronize(); CUDA_CHECK;
	norm_components <<< (unsigned)(w*h - 1) / 256 + 1, 256>>>(d_N, w*h, d_norm); CUDA_CHECK;
	hipDeviceSynchronize(); CUDA_CHECK;
	dim3 block(256, 1, 1);
	dim3 grid((unsigned)(w*h - 1) / block.x + 1, 3, 1);
	normalize_N <<< grid, block >>> (d_N, d_norm, w*h); CUDA_CHECK;
	return d_N;
}